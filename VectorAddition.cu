/*author: Zeke Elkins
 *date: 3/27/14
 *description: a CUDA program to add two vectors of integers
 */

#define N 512 //the size of the vector


#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

//device code to add two arrays of numbers;
//each block handles a different element of the array

__global__ void add(int *a, int *b, int *c) {
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

//method that will generate n random integers between 0 and 99
//values are stored in the int array a
void random_ints(int* a, int n) {

  int i;
  for (i=0; i<n; ++i) {
  a[i] = rand() % 100;
  }

}

int main(void) {

  int *a, *b, *c;       //host copies of a, b, c NOTE THESE ARE NOW POINTERS
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = N * sizeof(int); //size of N integers

  //allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  //allocate space for host copies of a, b, c and setup input vals
  a = (int *)malloc(size);
  random_ints(a, N);
  b = (int *)malloc(size);
  random_ints(b, N);
  c = (int *)malloc(size);

  //copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // launch add() kernel on GPU with N threads
  add<<<1,N>>>(d_a, d_b, d_c);

  //copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  //print out the results
  for (int i=0; i<N; i++) {
    cout << a[i] << " plus " << b[i] << " equals " << c[i] << endl;
  }

  //cleanup
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;

}