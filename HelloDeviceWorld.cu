/*author: Zeke Elkins
 *date: 3/27/2014
 *description: a simple hello world program -- introducing CUDA device syntax 
*/


#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void mykernel(void){
}

int main(void) {
 mykernel<<<1,1>>>(); 
 cout << "Hello World" << endl;
 return 0;
}

