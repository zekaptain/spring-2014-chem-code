/*author: Zeke Elkins
 *date: 3/27/14
 *description: a CUDA program to add two numbers
 */


#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

//DEVICE code
__global__ void add(int *a, int *b, int *c) {
  *c = *a + *b;
}

int main(void) {

  int a, b, c; //host copies of a, b, c
  int *d_a, *d_b, *d_c; //device copies of a, b, c
  int size = sizeof(int);

  //allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  //setup input values
  a = 2;
  b = 7;

  // copy inputs to device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  //launch add() kernel on GPU
  add<<<1,1>>>(d_a, d_b, d_c);

  //copy result back to host
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

  //output result
  cout << a << " plus " << b << " equals " << c << endl;
  
  //cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;

}